#include <algorithm>
#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

#define MOD 1000000007


__global__ void reset_cheapest(volatile int* cheapest,volatile int* cheapest2, int V) {
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < V) {
cheapest2[tid] = -1;
cheapest[tid] = 100000;
}
}

__global__ void reset_claim(volatile int* claim, int E) {
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < E) {
claim[tid] = 0;
}
}



__device__ int my_strcmp(const char* a, const char* b) {
int i = 0;
while (a[i]&&b[i] && a[i]==b[i]){
i++;
}
return a[i] - b[i];
}

__global__ void setter(int*d_w,char** d_type, int e){
int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid>=e) return;
if(my_strcmp(d_type[tid], "normal") == 0){
return;
}
else if(my_strcmp(d_type[tid], "dept") == 0){
d_w[tid]*=3;
}
else if(my_strcmp(d_type[tid], "green") == 0){
d_w[tid]*=2;
}
else{
d_w[tid]*=5;
}
}

__global__ void set2(volatile int* parent, volatile int* size,volatile int* cheapest,volatile int* cheap2,volatile int* claim,volatile int* mstw, volatile int* nc , int V) {
int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid>=V) return;
if(tid==0){
*mstw = 0;     
*nc = V; 
}
parent[tid] = tid;
size[tid] = 1;
cheapest[tid] = 100000;
cheap2[tid] = -1;
//claim[tid] = 0;
}


__device__ int find(volatile int* parent, int i) {

int root = i;
int current;
do{
current = root;
root = atomicAdd((int*)&parent[current], 0);  
}while (current != root);
    
// Path compression
while (i != root) {
int next = atomicAdd((int*)&parent[i], 0);
atomicExch((int*)&parent[i], root);   
i = next;
}
return root;

}

__global__ void cefev(volatile int* parent,int* s,int* d,int* w,volatile int* cheapest,int e) {

int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid>=e) return;    

int src = s[tid];
int dest = d[tid];
int weight =  w[tid];
    
int set1 = find(parent, src);
int set2 = find(parent, dest);

if(set1 != set2) {
if(cheapest[set1]>weight) {
atomicMin((int*)&cheapest[set1], weight);  
}
if(cheapest[set2]>weight) {
atomicMin((int*)&cheapest[set2], weight); 
}
}
}

__global__ void cefev2(volatile int* parent,int* s,int* d,int* w,volatile int* cheapest,volatile int* cheapest2,int e) {

int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid>=e) return;    

int src = s[tid];
int dest = d[tid];
int weight =  w[tid];
    
int set1 = find(parent, src);
int set2 = find(parent, dest);

if(set1 != set2) {
if(cheapest[set1]==weight) {
atomicExch((int*)&cheapest2[set1], tid);
}
if(cheapest[set2]==weight) {
atomicExch((int*)&cheapest2[set2], tid);
}
}
}


__global__ void mk(volatile int* parent,volatile int* size,int* s,int* d,int* w,volatile int* cheap2,volatile int* claim,volatile int* mst_weight,volatile int* num_comp,int V) {

int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= V) return;

int edge_id = cheap2[tid];
if (edge_id == -1) return;

if (atomicCAS((int*)&claim[edge_id], 0, 1) != 0) return;
int src = s[edge_id];
int dest = d[edge_id];
int weight = w[edge_id];

int set1 = find(parent, src);
int set2 = find(parent, dest);

if (set1 == set2) {
atomicExch((int*)&claim[edge_id], 0);
return;
}

int size1 = atomicAdd((int*)&size[set1], 0);
int size2 = atomicAdd((int*)&size[set2], 0);

bool merged = false;

if (size1 < size2) {
// Attempt to merge set1 into set2
int current_parent = atomicAdd((int*)&parent[set1], 0);
if (current_parent == set1 && atomicCAS((int*)&parent[set1], set1, set2) == set1) {
atomicAdd((int*)&size[set2], size1);
merged = true;
}
} 
else {
// Attempt to merge set2 into set1
int current_parent = atomicAdd((int*)&parent[set2], 0);
if (current_parent == set2 && atomicCAS((int*)&parent[set2], set2, set1) == set2) {
atomicAdd((int*)&size[set1], size2);
merged = true;
}
}
if (merged) {
atomicAdd((int*)mst_weight, weight);
atomicSub((int*)num_comp, 1);
} 
else{
atomicExch((int*)&claim[edge_id], 0);
 }
}
    


int main(){
 
int v, e;
cin >> v >> e;

vector<int> s;
vector<int> d;
vector<int> w;
vector<string> type;
vector<int> parent;
vector<int> size;

for(int i=0;i<e;i++){
int u,v,wt;
string t;
cin>>u>>v>>wt>>t;
s.push_back(u);
d.push_back(v);
w.push_back(wt);
type.push_back(t);
}
int mstw =0;
int* d_s;
int* d_d;
int* d_w;
char** d_type;
volatile int* d_parent;
volatile int* d_size;
volatile int* d_cheap;
volatile int* d_cheap2;
volatile int* d_claim;
volatile int* d_mstw;
volatile int* d_nc;
hipMalloc((void**)&d_parent, v*sizeof(int));
hipMalloc((void**)&d_size, v*sizeof(int));
hipMalloc((void**)&d_cheap, v*sizeof(int));
hipMalloc((void**)&d_cheap2, v*sizeof(int));
hipMalloc((void**)&d_claim, e*sizeof(int));
hipMalloc((void**)&d_mstw, sizeof(int));
hipMalloc((void**)&d_nc, sizeof(int));
hipMalloc(&d_s,e*sizeof(int));
hipMalloc(&d_d,e*sizeof(int));
hipMalloc(&d_w,e*sizeof(int));
hipMalloc(&d_type,e*sizeof(char*));
hipMemcpy(d_s, s.data(), e*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_d, d.data(), e*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_w, w.data(), e*sizeof(int), hipMemcpyHostToDevice);
char** h_type = new char*[e];
for(int i=0;i<e;i++) {
hipMalloc(&h_type[i], type[i].size() + 1);
hipMemcpy(h_type[i], type[i].c_str(), type[i].size() + 1, hipMemcpyHostToDevice);
}
hipMemcpy(d_type, h_type, e * sizeof(char*), hipMemcpyHostToDevice);

int psb = ceil(e/1024.0);
int s2b = ceil(v/1024.0);
int nc = v;

auto start = std::chrono::high_resolution_clock::now(); // keep it just before the kernel launch
set2<<<s2b,1024>>>(d_parent, d_size,d_cheap,d_cheap2,d_claim,d_mstw,d_nc,v);
setter<<<psb,1024>>>(d_w,d_type,e);
//cudaDeviceSynchronize();
while(nc>1){
reset_cheapest<<<s2b, 1024>>>(d_cheap,d_cheap2, v);
reset_claim<<<psb, 1024>>>(d_claim, e);
//cudaDeviceSynchronize();
cefev<<<psb,1024>>>(d_parent,d_s,d_d,d_w,d_cheap,e);
//cudaDeviceSynchronize();
cefev2<<<psb,1024>>>(d_parent,d_s,d_d,d_w,d_cheap,d_cheap2,e);
//cudaDeviceSynchronize();
mk<<<s2b,1024>>>(d_parent,d_size,d_s,d_d,d_w,d_cheap2,d_claim,d_mstw,d_nc,v);
//cudaDeviceSynchronize();
hipMemcpy(&nc,(void*)d_nc,sizeof(int),hipMemcpyDeviceToHost);
}
hipDeviceSynchronize();
auto end = std::chrono::high_resolution_clock::now(); // keep it just after the kernel launch
hipMemcpy(&mstw,(void*)d_mstw,sizeof(int),hipMemcpyDeviceToHost);
cout<<mstw%MOD<<"\n";
std::chrono::duration<double> elapsed1 = end - start;
//cout<< elapsed1.count() << "\n";
for (int i = 0; i < e; i++) {
hipFree(h_type[i]);
}    
delete[] h_type;
hipFree(d_s);
hipFree(d_d);
hipFree(d_w);
hipFree(d_type);
hipFree((void*)d_parent);
hipFree((void*)d_size);
hipFree((void*)d_cheap);
hipFree((void*)d_cheap2);
hipFree((void*)d_claim);
hipFree((void*)d_mstw);
hipFree((void*)d_nc);

return 0;
}

